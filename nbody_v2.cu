#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "timer.h" // Include the timer header
#include "matric.h" // Include your custom matric.h header

#define SOFTENING 1e-9f

typedef struct {
    float x, y, z, vx, vy, vz;
} Body;

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__ void bodyForce(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;

        for (int j = 0; j < n; j++) {
            if (i != j) {
                float dx = p[j].x - p[i].x;
                float dy = p[j].y - p[i].y;
                float dz = p[j].z - p[i].z;
                float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
                float invDist = rsqrtf(distSqr);
                float invDist3 = invDist * invDist * invDist;

                Fx += dx * invDist3;
                Fy += dy * invDist3;
                Fz += dz * invDist3;
            }
        }

        p[i].vx += dt * Fx;
        p[i].vy += dt * Fy;
        p[i].vz += dt * Fz;
    }
}

__global__ void integratePositions(Body *p, float dt, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        p[i].x += p[i].vx * dt;
        p[i].y += p[i].vy * dt;
        p[i].z += p[i].vz * dt;
    }
}

void saveForcesToFile(const char *filename, int nBodies, Body *p) {
    FILE *file = fopen(filename, "w");
    if (!file) {
        fprintf(stderr, "Unable to open file %s for writing.\n", filename);
        return;
    }
    for (int i = 0; i < nBodies; i++) {
        fprintf(file, "Body %d: x = %.3f, y = %.3f, z = %.3f, vx = %.3f, vy = %.3f, vz = %.3f\n",
                i, p[i].x, p[i].y, p[i].z, p[i].vx, p[i].vy, p[i].vz);
    }
    fclose(file);
}

int main(int argc, char **argv) {
    int nBodies = 200000;
    if (argc > 1) nBodies = atoi(argv[1]);

    const float dt = 0.01f; // time step
    const int nIters = 500;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    Body *p = (Body *)malloc(bytes);

    if (p == NULL) {
        fprintf(stderr, "Unable to allocate memory for bodies.\n");
        return 1;
    }

    float *buf = (float *)malloc(6 * nBodies * sizeof(float));
    if (buf == NULL) {
        fprintf(stderr, "Unable to allocate memory for buffer.\n");
        free(p);
        return 1;
    }

    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data
    for (int i = 0; i < nBodies; i++) {
        p[i].x = buf[6 * i];
        p[i].y = buf[6 * i + 1];
        p[i].z = buf[6 * i + 2];
        p[i].vx = buf[6 * i + 3];
        p[i].vy = buf[6 * i + 4];
        p[i].vz = buf[6 * i + 5];
    }

    free(buf);

    // Allocate device memory
    Body *d_p;
    hipMalloc(&d_p, bytes);

    hipMemcpy(d_p, p, bytes, hipMemcpyHostToDevice);

    double totalTime = 0.0;

    int blockSize = 256; // adjust value for performance tuning
    int numBlocks = (nBodies + blockSize - 1) / blockSize;

    for (int iter = 1; iter <= nIters; iter++) {
        StartTimer();

        bodyForce<<<numBlocks, blockSize>>>(d_p, dt, nBodies);
        hipDeviceSynchronize();
        integratePositions<<<numBlocks, blockSize>>>(d_p, dt, nBodies);
        hipDeviceSynchronize();

        const double tElapsed = GetTimer() / 1000.0;
        if (iter > 1) { // First iter is warm up
            totalTime += tElapsed;
        }
        printf("Iteration %d: %.3f seconds\n", iter, tElapsed);
    }

    hipMemcpy(p, d_p, bytes, hipMemcpyDeviceToHost);

    saveForcesToFile("forces.txt", nBodies, p);

    double avgTime = totalTime / (double)(nIters - 1);
    double rate = (double)nBodies / avgTime;

    printf("Average rate for iterations 2 through %d: %.3f steps per second.\n",
           nIters, rate);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);

    free(p);

    hipFree(d_p);

    return 0;
}
